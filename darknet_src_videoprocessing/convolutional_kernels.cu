#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")  
#endif

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += abs(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += abs(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

__global__ void cuda_f32_to_f16(float* input_f32, size_t size, half *output_f16)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) output_f16[idx] = __float2half(input_f32[idx]);
	//if (idx < size) *((unsigned short *)output_f16 + idx) = __float2half(input_f32[idx]);
}

void cuda_convert_f32_to_f16(float* input_f32, size_t size, half *output_f16) {
	cuda_f32_to_f16 <<< size / BLOCK + 1, BLOCK, 0, get_cuda_stream() >>> (input_f32, size, output_f16);
}

__global__ void cuda_f16_to_f32(half* input_f16, size_t size, float *output_f32)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) output_f32[idx] = __half2float(input_f16[idx]);
	//if (idx < size) output_f32[idx] = __half2float(*((unsigned short *)input_f16 + idx));
}

void cuda_convert_f16_to_f32(half* input_f16, size_t size, float *output_f32) {
	cuda_f16_to_f32 <<< size / BLOCK + 1, BLOCK, 0, get_cuda_stream() >>> (input_f16, size, output_f32);
}

half *cuda_make_f16_from_f32_array(float *src, size_t n)
{
	half *dst16;
	size_t size = sizeof(half)*n;
	check_error(hipMalloc((void **)&dst16, size));
	if (src) {
		cuda_convert_f32_to_f16(src, n, dst16);
	}
	if (!dst16) error("Cuda malloc failed\n");
	return dst16;
}

void forward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(state.input, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        state.input = l.binary_input_gpu;
    }

#ifdef CUDNN
	//float one = 1;	// alpha[0], beta[0] is float for HALF and FLOAT
	float alpha = 1, beta = 0;

#ifdef CUDNN_HALF
	// Note: For improved performance it is advised to use beta[0] = 0.0. 
	// For Tensor Core: hipdnnSetConvolutionMathType() where hipdnnMathType_t mathType = HIPDNN_TENSOR_OP_MATH;
	// 1. or HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM and use HIPDNN_DATA_HALF
	// 2. or HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED
	// More: http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#tensor_ops

	const size_t input16_size = l.batch*l.c*l.w*l.h;
	static size_t max_input16_size = input16_size;
	static half* input16 = cuda_make_f16_from_f32_array(NULL, max_input16_size);

	const size_t output16_size = l.batch*l.out_c*l.out_h*l.out_w;
	static size_t max_output16_size = output16_size;
	static half* output16 = cuda_make_f16_from_f32_array(NULL, max_output16_size);

	if (max_input16_size < input16_size) {
		max_input16_size = input16_size;
		cuda_free((float *)input16);
		input16 = cuda_make_f16_from_f32_array(state.input, max_input16_size);
	}

	if (max_output16_size < output16_size) {
		max_output16_size = output16_size;
		cuda_free((float *)output16);
		output16 = cuda_make_f16_from_f32_array(NULL, max_output16_size);
	}

	cuda_convert_f32_to_f16(state.input, input16_size, input16);

	hipdnnConvolutionForward(cudnn_handle(),
		&alpha,
		l.srcTensorDesc,
		input16,
		l.weightDesc,
		l.weights_gpu16,
		l.convDesc,
		l.fw_algo,
		state.workspace,
		l.workspace_size,
		&beta,
		l.dstTensorDesc,
		output16);
	
	cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
#else

    hipdnnConvolutionForward(cudnn_handle(),
                &alpha,
                l.srcTensorDesc,
                state.input,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                state.workspace,
                l.workspace_size,
                &beta,
                l.dstTensorDesc,
                l.output_gpu);
#endif


#else
    int i;
    int m = l.n;
    int k = l.size*l.size*l.c;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        float * a = l.weights_gpu;
        float * b = state.workspace;
        float * c = l.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }
#endif

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, state);
    }
    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);

    activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
	//hipDeviceSynchronize();	// for correct profiling of performance
}

void backward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, state);
        //axpy_ongpu(l.outputs*l.batch, -state.net.decay, l.x_gpu, 1, l.delta_gpu, 1);
    } else {
        //axpy_ongpu(l.outputs*l.batch, -state.net.decay, l.output_gpu, 1, l.delta_gpu, 1);
    }
    float *original_input = state.input;

    if(l.xnor) state.input = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            state.input,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            state.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(state.delta){
        if(l.binary || l.xnor) swap_binary(&l);
		// http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                state.delta);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
    }

#else
    int m = l.n;
    int n = l.size*l.size*l.c;
    int k = l.out_w*l.out_h;

    int i;
    for(i = 0; i < l.batch; ++i){
        float * a = l.delta_gpu;
        float * b = state.workspace;
        float * c = l.weight_updates_gpu;

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(state.delta){
            if(l.binary || l.xnor) swap_binary(&l);
            float * a = l.weights_gpu;
            float * b = l.delta_gpu;
            float * c = state.workspace;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(state.workspace, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.delta + i*l.c*l.h*l.w);
            if(l.binary || l.xnor) {
                swap_binary(&l);
            }
            if(l.xnor) gradient_array_ongpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, state.delta + i*l.c*l.h*l.w);
        }
    }
#endif
}

void pull_convolutional_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
    if (layer.adam){
        cuda_pull_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_pull_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
}

void push_convolutional_layer(convolutional_layer layer)
{
    cuda_push_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
#ifdef CUDNN_HALF
	cuda_convert_f32_to_f16(layer.weights_gpu, layer.c*layer.n*layer.size*layer.size, (half *)layer.weights_gpu16);
#endif
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
    if (layer.adam){
        cuda_push_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_push_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
}

void update_convolutional_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;
    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    if(layer.scales_gpu){
        axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
        scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);
    }

    if(layer.adam){
        scal_ongpu(size, layer.B1, layer.m_gpu, 1);
        scal_ongpu(size, layer.B2, layer.v_gpu, 1);

        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);

        axpy_ongpu(size, -(1-layer.B1), layer.weight_updates_gpu, 1, layer.m_gpu, 1);
        mul_ongpu(size, layer.weight_updates_gpu, 1, layer.weight_updates_gpu, 1);
        axpy_ongpu(size, (1-layer.B2), layer.weight_updates_gpu, 1, layer.v_gpu, 1);

        adam_gpu(size, layer.weights_gpu, layer.m_gpu, layer.v_gpu, layer.B1, layer.B2, learning_rate/batch, layer.eps, layer.t+1);
        fill_ongpu(size, 0, layer.weight_updates_gpu, 1);
    }else{
        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);
        axpy_ongpu(size, learning_rate/batch, layer.weight_updates_gpu, 1, layer.weights_gpu, 1);
        scal_ongpu(size, momentum, layer.weight_updates_gpu, 1);
    }
}


